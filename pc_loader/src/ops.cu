#include "hip/hip_runtime.h"
#include "types_gpu.hpp"

// Kernel definition
__global__ void computeRange(const float4* point_cloud, float* ranges, size_t num_current_points) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    
    if (idx < num_current_points) {
        float range = 0;
        for(int i=0; i < NUM_FIELDS - 1; i++) {
            auto coord = point_cloud[idx * NUM_FIELDS + i];
            range += coord * coord;
        }
        ranges[idx] = std::sqrt(range);
    }
    
}

void computeRangeKernel(const float* point_cloud_gpu_raw_data, float* range_array_gpu, size_t num_current_points) {
    int inputSize = num_current_points; // The size of the input data
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);
    int threadsPerBlock = props.maxThreadsPerBlock;

    int blocksPerGrid = ((inputSize + threadsPerBlock - 1) / threadsPerBlock);
        
    computeRange<<<blocksPerGrid, threadsPerBlock>>>(point_cloud_gpu_raw_data, range_array_gpu, num_current_points);
    gpuErrchk( hipPeekAtLastError() );
}
